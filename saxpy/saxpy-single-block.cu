#include <iostream>
#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void saxpy(double *z, double *x, double *y, double alpha, size_t N) {

    int i = threadIdx.x;
    z[i] = alpha*x[i] + y[i];
}

int main() {

    double *h_z, *h_x, *h_y;
    double *d_z, *d_x, *d_y;
    double alpha = 1.5;
    const size_t N = 100;

    h_z = new double[N];
    h_x = new double[N];
    h_y = new double[N];

    for (size_t i = 0; i < N; i += 1) {
        h_x[i] = 5.0;
        h_y[i] = -2.0;
    }

    hipMalloc(&d_z, N*sizeof(double));
    hipMalloc(&d_x, N*sizeof(double));
    hipMalloc(&d_y, N*sizeof(double));

    hipMemcpy(d_x, h_x, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N*sizeof(double), hipMemcpyHostToDevice);

    delete[] h_x;
    delete[] h_y;

    
    saxpy<<<1, N>>>(d_z, d_x, d_y, alpha, N);


    hipMemcpy(h_z, d_z, N*sizeof(double), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < N; i += 1) {
        std::cout << h_z[i] << " ";
    }
    std::cout << "\n";

    delete[] h_z;

    hipFree(d_z);
    hipFree(d_x);
    hipFree(d_y);
}
