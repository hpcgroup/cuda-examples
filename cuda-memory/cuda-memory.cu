#include <iostream>
#include <cstdint>
#include <cstdlib>
#include <hip/hip_runtime.h>


int main() {

    double *h_z, *h_x, *h_y;
    double *d_z, *d_x, *d_y;
    const size_t N = 100;

    h_z = new double[N];
    h_x = new double[N];
    h_y = new double[N];

    for (size_t i = 0; i < N; i += 1) {
        h_x[i] = rand();
        h_y[i] = 1.0 / (h_x[i] + 1E8);
    }

    hipMalloc(&d_z, N);
    hipMalloc(&d_x, N);
    hipMalloc(&d_y, N);

    hipMemcpy(d_x, h_x, N, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N, hipMemcpyHostToDevice);
    delete[] h_x;
    delete[] h_y;

    
    /* ... */
    

    hipMemcpy(h_z, d_z, N, hipMemcpyDeviceToHost);

    for (size_t i = 0; i < N; i += 1) {
        std::cout << h_z[i] << " ";
    }
    std::cout << "\n";

    delete[] h_z;

    hipFree(d_z);
    hipFree(d_x);
    hipFree(d_y);
}
