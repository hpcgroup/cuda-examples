#include <iostream>
#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>

/** Compute C = A.B + C
 *  where A is MxP, B is PxN, and C is MxN
 */
__global__ void matmul(double *C, double *A, double *B, size_t M, size_t P, size_t N) {

    int i = blockDim.x*blockIdx.x + threadIdx.x;
    int j = blockDim.y*blockIdx.y + threadIdx.y;

    if (i < M && j < N) {
        for (int k = 0; k < P; k += 1) {
            C[i*N+j] += A[i*P+k]*B[k*N+j];
        }
    }
}

int main() {

    double *h_C, *h_A, *h_B;
    double *d_C, *d_A, *d_B;
    const size_t M = 1024, P = 1024, N = 1024;

    h_C = new double[M*N];
    h_A = new double[M*P];
    h_B = new double[P*N];

    for (size_t i = 0; i < M; i += 1) {
        for (size_t j = 0; j < P; j += 1) {
            h_A[i*P+j] = (i==j) ? 1.0 : 0.0;
        }
    }
    for (size_t i = 0; i < P*N; i += 1) {
        h_B[i] = 42.0;
    }

    hipMalloc(&d_C, M*N*sizeof(double));
    hipMalloc(&d_A, M*P*sizeof(double));
    hipMalloc(&d_B, P*N*sizeof(double));

    hipMemcpy(d_A, h_A, M*P*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, P*N*sizeof(double), hipMemcpyHostToDevice);

    delete[] h_A;
    delete[] h_B;

    dim3 threadsPerBlock (16, 16);
    dim3 numBlocks (
        M/threadsPerBlock.x + (M%threadsPerBlock.x != 0),
        N/threadsPerBlock.y + (N%threadsPerBlock.y != 0));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    matmul<<<numBlocks, threadsPerBlock>>>(d_C, d_A, d_B, M, P, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Elapsed time = %f\n", elapsed);


    hipMemcpy(h_C, d_C, M*N*sizeof(double), hipMemcpyDeviceToHost);

    bool success = true;
    for (size_t i = 0; i < M*N; i += 1) {
        if (std::abs(h_C[i] - (42.0)) > 1E-8)
            success = false;
    }
    if (!success)
        fprintf(stderr, "Error: incorrect results!\n");

    delete[] h_C;

    hipFree(d_C);
    hipFree(d_A);
    hipFree(d_B);
}
